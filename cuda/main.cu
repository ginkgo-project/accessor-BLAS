#include "hip/hip_runtime.h"
#include <array>
#include <ios>
#include <iostream>
#include <random>
#include <stdexcept>
#include <type_traits>
#include <vector>
#include <limits>
#include <cmath>

#include "kernels.cuh"
#include "utils.cuh"

template <typename ValueType>
void control_spmv(const matrix_info minfo, const std::vector<ValueType> &mtx,
                  const matrix_info vinfo, const std::vector<ValueType> &b,
                  std::vector<ValueType> &res) {
    for (std::size_t row = 0; row < minfo.size[0]; ++row) {
        for (std::size_t col = 0; col < minfo.size[1]; ++col) {
            const std::size_t midx = row * minfo.stride + col;
            res[row] += mtx[midx] * b[col];
        }
    }
}

template <typename ValueType>
void compare(const matrix_info info, const std::vector<ValueType> &mtx1,
             const std::vector<ValueType> &mtx2) {
    for (std::size_t row = 0; row < info.size[0]; ++row) {
        for (std::size_t col = 0; col < info.size[1]; ++col) {
            const std::size_t midx = row * info.stride + col;
            const auto v1 = mtx1[midx];
            const auto v2 = mtx2[midx];
            if (std::abs(v1 - v2) > std::numeric_limits<ValueType>::epsilon() * 8) {
                std::cout << "Mismatch at (" << row << ", " << col << "): " << v1 << " vs. " << v2 << '\n';
            }
        }
    }
}

template <typename t1, typename t2, typename t3, typename t4, typename t5>
void test(t1, t2, t3, t4, t5) {
    static_assert(std::is_same<t1, t2>::value,
                  "GKO_DECLARE_OUTPLACE_ABSOLUTE_DENSE_KERNEL");
}

int main() {
    using vtype = double;
    using stype = float;
    constexpr int bench_iters {10};
    constexpr std::int32_t num_rows{16 * 1024};
    constexpr matrix_info minfo{{num_rows, num_rows}};
    
    std::cout << "Total matrix size: " << minfo.get_1d_size() * sizeof(vtype) << '\n';
    static_assert(minfo.size[0] == minfo.size[1], "Matrix must be square!");
    
    std::default_random_engine rengine(42);
    std::uniform_real_distribution<stype> val_dist(1.0, 2.0);
    std::uniform_real_distribution<stype> one_dist(1.0, 1.0);

    const matrix_info vinfo{{minfo.size[0], 1}};

    auto v_matrix = gen_mtx<vtype>(minfo, val_dist, rengine);
    auto s_matrix = convert_mtx<stype>(minfo, v_matrix);

    auto v_b = gen_mtx<vtype>(vinfo, one_dist, rengine);
    auto s_b = convert_mtx<stype>(vinfo, v_matrix);

    auto v_res = std::vector<vtype>(vinfo.get_1d_size(), vtype{});
    auto s_res = std::vector<stype>(vinfo.get_1d_size(), stype{});

    auto dv_matrix = GpuMemory<vtype>(minfo.get_1d_size());
    dv_matrix.copy_from(v_matrix);
    auto ds_matrix = GpuMemory<stype>(minfo.get_1d_size());
    ds_matrix.copy_from(s_matrix);

    auto dv_b = GpuMemory<vtype>(vinfo.get_1d_size());
    dv_b.copy_from(v_b);
    auto ds_b = GpuMemory<stype>(vinfo.get_1d_size());
    ds_b.copy_from(s_b);
    auto dv_res = GpuMemory<vtype>(vinfo.get_1d_size());
    auto ds_res = GpuMemory<stype>(vinfo.get_1d_size());

    CudaTimer ctimer;
    // Benchmark double
    // Warmup
    spmv(minfo, dv_matrix.data(), vinfo, dv_b.data(), dv_res.data());
    synchronize();
    
    double double_time{};
    for (int i = 0; i < bench_iters; ++i) {
        ctimer.start();
        spmv(minfo, dv_matrix.data(), vinfo, dv_b.data(), dv_res.data());
        ctimer.stop();
        double_time += ctimer.get_time();
        ctimer.reset();
    }
    std::cout << "double time: " << double_time / bench_iters << " ms\n";
    
    // Benchmark float
    // Warmup
    spmv(minfo, ds_matrix.data(), vinfo, ds_b.data(), ds_res.data());
    synchronize();
    
    double single_time{};
    for (int i = 0; i < bench_iters; ++i) {
        ctimer.start();
        spmv(minfo, ds_matrix.data(), vinfo, ds_b.data(), ds_res.data());
        ctimer.stop();
        single_time += ctimer.get_time();
        ctimer.reset();
    }
    std::cout << "single time: " << single_time / bench_iters << " ms\n";
    
    // Benchmark Accessor<fp64, fp32>
    // Warmup
    acc_spmv<vtype>(minfo, ds_matrix.data(), vinfo, ds_b.data(), ds_res.data());
    synchronize();
    
    double acc_time{};
    for (int i = 0; i < bench_iters; ++i) {
        ctimer.start();
        acc_spmv<vtype>(minfo, ds_matrix.data(), vinfo, ds_b.data(), ds_res.data());
        ctimer.stop();
        acc_time += ctimer.get_time();
        ctimer.reset();
    }
    std::cout << "Access time: " << acc_time / bench_iters << " ms\n";
    
    auto gpu_res = dv_res.get_vector();

    // Control impl:
    control_spmv(minfo, v_matrix, vinfo, v_b, v_res);

    std::cout.precision(16);
    std::cout << std::scientific;

    // print_mtx(vinfo, v_res);
    std::cout << "Comparison:\n";
    // print_mtx(vinfo, gpu_res);
    compare(vinfo, v_res, gpu_res);

}
