#include "hip/hip_runtime.h"
#include <array>
#include <cmath>
#include <ios>
#include <iostream>
#include <limits>
#include <random>
#include <stdexcept>
#include <type_traits>
#include <vector>

#include "../error_tobias.hpp"
#include "kernels.cuh"
#include "utils.cuh"

// TODO properly use alpha and beta for GEMV
template <typename ValueType>
void control_gemv(const matrix_info minfo, ValueType alpha,
                  const std::vector<ValueType> &mtx, const matrix_info vinfo,
                  ValueType beta, const std::vector<ValueType> &x,
                  std::vector<ValueType> &res) {
    if (vinfo.size[1] != 1) {
        throw "Error!";
    }
    for (std::size_t i = 0; i < vinfo.size[0]; ++i) {
        res[i * vinfo.stride] *= beta;
    }
    for (std::size_t row = 0; row < minfo.size[0]; ++row) {
        for (std::size_t col = 0; col < minfo.size[1]; ++col) {
            const std::size_t midx = row * minfo.stride + col;
            res[row] += alpha * mtx[midx] * x[col * vinfo.stride];
        }
    }
}

template <typename OutputType, typename VectorType, typename ReduceOp>
OutputType reduce(const matrix_info info, VectorType &tmp, ReduceOp op) {
    std::size_t end = info.size[0];
    for (std::size_t halfway = ceildiv(info.size[0], std::size_t{2});
         halfway > 1; halfway = ceildiv(halfway, std::size_t{2})) {
        for (std::size_t row = 0; row < halfway; ++row) {
            if (row + halfway < end) {
                for (std::size_t col = 0; col < info.size[1]; ++col) {
                    const std::size_t midx = row * info.stride + col;
                    const std::size_t midx2 =
                        (row + halfway) * info.stride + col;
                    tmp[midx] = op(tmp[midx], tmp[midx2]);
                }
            }
        }
        end = halfway;
    }
    return static_cast<OutputType>(info.size[0] == 1 ? op(tmp[0], {})
                                                     : op(tmp[0], tmp[1]));
}

template <typename T>
std::enable_if_t<std::is_floating_point<T>::value, T> get_value(T val) {
    return val;
}

template <typename T>
std::enable_if_t<!std::is_floating_point<T>::value, typename T::value_type>
get_value(T val) {
    return val.e;
}

template <typename ReferenceType, typename OtherType, typename ValueType>
ValueType compare(const matrix_info info,
                  const std::vector<ReferenceType> &mtx1,
                  const std::vector<OtherType> &mtx2,
                  std::vector<ValueType> &tmp) {
    // ReferenceType error{};
    using return_type = decltype(get_value(ReferenceType{}));
    if (info.get_1d_size() > mtx1.size() || info.get_1d_size() > mtx2.size() ||
        info.get_1d_size() > tmp.size() || info.size[1] != 1) {
        throw "Error";
    }
    for (std::size_t row = 0; row < info.size[0]; ++row) {
        const std::size_t midx = row * info.stride;
        tmp[midx] = ValueType{};
    }
    for (std::size_t row = 0; row < info.size[0]; ++row) {
        const std::size_t midx = row * info.stride;
        const auto v1 = get_value(mtx1[midx]);
        const decltype(v1) v2 = get_value(mtx2[midx]);
        const auto delta = std::abs(v1 - v2);
        tmp[midx] = delta;
    }
    /*
    std::cout << '\n';
    for (std::size_t i = 0; i < info.size[0]; ++i) {
        std::cout << tmp[i] << ' ';
    }
    std::cout << '\n';
    for (std::size_t i = 0; i < info.size[0]; ++i) {
        std::cout << i % 10 << ' ';
        for (int j = 10; j < tmp[i]; j *= 10)
            std::cout << ' ';
    }
    std::cout << '\n';
    //*/
    return reduce<ValueType>(
        info, tmp, [](ValueType o1, ValueType o2) { return o1 + o2; });
}

template <typename Callable>
double benchmark_function(Callable func) {
    constexpr int bench_iters{10};
    double time_ms[bench_iters];
    CudaTimer ctimer;
    // Warmup
    func();
    synchronize();
    for (int i = 0; i < bench_iters; ++i) {
        ctimer.start();
        func();
        ctimer.stop();
        time_ms[i] = ctimer.get_time();
        ctimer.reset();
    }

    // Reduce timings to one value
    double result_ms{std::numeric_limits<double>::max()};
    for (int i = 0; i < bench_iters; ++i) {
        result_ms = std::min(result_ms, time_ms[i]);
    }
    // result_ms /= static_cast<double>(bench_iters);
    return bench_iters == 0 ? double{} : result_ms;
}

template <typename ValueType>
class GemvMemory {
   public:
    template <typename MtxDist, typename VectDist, typename RndEngine>
    GemvMemory(std::size_t max_rows, std::size_t max_cols, MtxDist &&mtx_dist,
               VectDist &&vect_dist, RndEngine &&engine)
        : m_info_{{max_rows, max_cols}},
          x_info_{{max_cols, 1}},
          res_info_{{max_rows, 1}},
          cpu_mtx_(gen_mtx(m_info_, mtx_dist, engine)),
          cpu_x_(gen_mtx(x_info_, vect_dist, engine)),
          cpu_res_(gen_mtx(res_info_, vect_dist, engine)),
          gpu_mtx_(m_info_.get_1d_size()),
          gpu_x_(x_info_.get_1d_size()),
          gpu_res_(res_info_.get_1d_size())
    {
        gpu_mtx_.copy_from(cpu_mtx_);
        gpu_x_.copy_from(cpu_x_);
        gpu_res_.copy_from(cpu_res_);
    }
    template <typename OtherType>
    GemvMemory(const GemvMemory<OtherType> &other)
        : m_info_(other.m_info_),
          x_info_(other.x_info_),
          res_info_(other.res_info_),
          cpu_mtx_(m_info_.get_1d_size()),
          cpu_x_(x_info_.get_1d_size()),
          cpu_res_(res_info_.get_1d_size()),
          gpu_mtx_(m_info_.get_1d_size()),
          gpu_x_(x_info_.get_1d_size()),
          gpu_res_(res_info_.get_1d_size())
    {
        convert_mtx(m_info_, other.cpu_mtx_, cpu_mtx_,
                    [](OtherType v) { return static_cast<ValueType>(v); });
        convert_mtx(x_info_, other.cpu_x_, cpu_x_,
                    [](OtherType v) { return static_cast<ValueType>(v); });
        convert_mtx(res_info_, other.cpu_res_, cpu_res_,
                    [](OtherType v) { return static_cast<ValueType>(v); });
        gpu_mtx_.copy_from(cpu_mtx_);
        gpu_x_.copy_from(cpu_x_);
        gpu_res_.copy_from(cpu_res_);
    }

   private:
    const matrix_info m_info_;
    const matrix_info x_info_;
    const matrix_info res_info_;

    std::vector<ValueType> cpu_mtx_;
    std::vector<ValueType> cpu_x_;
    std::vector<ValueType> cpu_res_;

    GpuMemory<ValueType> gpu_mtx_;
    GpuMemory<ValueType> gpu_x_;
    GpuMemory<ValueType> gpu_res_;
};

int main() {
    /*
    using ar_type = error_number<double>;
    using st_type = error_number<float>;
    using value_type = ar_type::value_type;
    auto convert_func = [](ar_type val) {
        return st_type{static_cast<st_type::value_type>(val.v),
                       static_cast<st_type::value_type>(val.e)};
    };
    /*/
    using ar_type = double;
    using st_type = float;
    using value_type = ar_type;
    auto convert_func = [](ar_type val) { return static_cast<st_type>(val); };

    //*/

    constexpr std::size_t max_rows{24 * 1024};
    constexpr matrix_info max_minfo{{max_rows, max_rows}};
    constexpr char DELIM{';'};

    const ar_type aalpha{1.0};
    const ar_type abeta{1.0};
    const st_type salpha{static_cast<st_type>(aalpha)};
    const st_type sbeta{static_cast<st_type>(abeta)};
    std::default_random_engine rengine(42);
    std::uniform_real_distribution<value_type> mtx_dist(-2.0, 2.0);
    // std::normal_distribution<value_type> mtx_dist(1, 2);
    //*
    // std::uniform_real_distribution<value_type> vector_dist(-2.0, 2.0);
    // std::uniform_real_distribution<value_type> vector_dist(1.0, 1.0);
    auto vector_dist = mtx_dist;
    /*/

    auto vector_dist = [rnd = 0](auto val) mutable {
        return (rnd = (rnd + 1) % 40) == 0
                   ? 1
                   : std::numeric_limits<float>::epsilon() / 2;
        // return std::numeric_limits<float>::epsilon() / 2;
    };
    std::cout << std::numeric_limits<float>::epsilon() / 2 << '\n';
    //*/

    /*
    constexpr std::size_t tmp_size{1000};
    std::vector<float> tmp(tmp_size);
    for (int i = 0; i < tmp_size; ++i) {
        tmp[i] = vector_dist(1);
    }

    double d_sum{};
    float f_sum{};
    for (int i = 0; i < tmp_size; ++i) {
        d_sum += tmp[i];
        f_sum += tmp[i];
    }
    std::cout.precision(16);
    std::cout << std::scientific;
    std::cout << "float  sum = " << f_sum
            << "\ndouble sum = " << static_cast<float>(d_sum)
            << '\n';
    return 0;
    //*/
    /*
    constexpr std::size_t red_size{1};
    std::cout << "Beginning...\n";
    std::vector<int> a(red_size, 0);
    std::vector<int> b(red_size, 1);
    std::vector<int> tmp(red_size);
    std::cout << "Running compare...\n";
    std::cout << compare({{red_size, 1}}, a, b, tmp) << '\n';
    return 0;
    //*/

    const matrix_info max_vinfo{{max_rows, 1}};

    auto v_matrix = gen_mtx<ar_type>(max_minfo, mtx_dist, rengine);
    std::vector<st_type> s_matrix(max_minfo.get_1d_size());
    convert_mtx<st_type>(max_minfo, v_matrix, s_matrix, convert_func);

    auto v_b = gen_mtx<ar_type>(max_vinfo, vector_dist, rengine);
    std::vector<st_type> s_b(max_vinfo.get_1d_size());
    convert_mtx<st_type>(max_vinfo, v_b, s_b, convert_func);

    auto v_res = std::vector<ar_type>(max_vinfo.get_1d_size(), ar_type{});
    auto v_res_ref = std::vector<ar_type>(max_vinfo.get_1d_size(), ar_type{});
    auto s_res = std::vector<st_type>(max_vinfo.get_1d_size(), st_type{});
    auto v_reduce =
        std::vector<value_type>(max_vinfo.get_1d_size(), value_type{});

    auto dv_matrix = GpuMemory<ar_type>(max_minfo.get_1d_size());
    dv_matrix.copy_from(v_matrix);
    auto ds_matrix = GpuMemory<st_type>(max_minfo.get_1d_size());
    ds_matrix.copy_from(s_matrix);

    auto dv_b = GpuMemory<ar_type>(max_vinfo.get_1d_size());
    dv_b.copy_from(v_b);
    auto ds_b = GpuMemory<st_type>(max_vinfo.get_1d_size());
    ds_b.copy_from(s_b);
    auto dv_res = GpuMemory<ar_type>(max_vinfo.get_1d_size());
    auto ds_res = GpuMemory<st_type>(max_vinfo.get_1d_size());
    dv_res.copy_from(v_res);
    ds_res.copy_from(s_res);

    auto cublasHandle = get_cublas_handle();

    std::cout << "Num Rows" << DELIM << "GEMV double" << DELIM << "GEMV float"
              << DELIM << "GEMV Acc<fp64, fp64>" << DELIM
              << "GEMV Acc<fp64, fp32>" << DELIM << "CUBLAS GEMV fp64" << DELIM
              << "CUBLAS GEMV fp32" << '\n';

    std::cout.precision(16);
    std::cout << std::scientific;
    /*
    std::cout << "single_error           Acc<fp64, fp32> error    single_error "
                 "/ acc_error\n";
    */

    constexpr auto start = max_rows / 48;
    constexpr auto row_incr = start;
    for (auto num_rows = start; num_rows <= max_rows; num_rows += row_incr) {
        // for (int i = 0; i < 10; ++i) {
        const matrix_info minfo{{num_rows, num_rows}};
        const matrix_info vinfo{{num_rows, 1}};

        v_matrix = gen_mtx<ar_type>(minfo, mtx_dist, rengine);
        convert_mtx<st_type>(minfo, v_matrix, s_matrix, convert_func);
        dv_matrix.copy_from(v_matrix);
        ds_matrix.copy_from(s_matrix);

        double d_time{};
        auto d_func = [&]() {
            gemv(minfo, aalpha, dv_matrix.data(), vinfo, dv_b.data(), abeta,
                 dv_res.data());
        };
        double s_time{};
        auto s_func = [&]() {
            gemv(minfo, salpha, ds_matrix.data(), vinfo, ds_b.data(), sbeta,
                 ds_res.data());
        };
        double avv_time{};
        auto avv_func = [&]() {
            acc_gemv<ar_type>(minfo, aalpha, dv_matrix.data(), vinfo,
                              dv_b.data(), abeta, dv_res.data());
        };
        double avs_time{};
        auto avs_func = [&]() {
            acc_gemv<ar_type>(minfo, aalpha, ds_matrix.data(), vinfo,
                              ds_b.data(), abeta, ds_res.data());
        };
        double cd_time{};
        auto cd_func = [&]() {
            cublas_gemv(cublasHandle.get(), minfo, aalpha, dv_matrix.data(),
                        vinfo, dv_b.data(), abeta, dv_res.data());
        };
        double cs_time{};
        auto cs_func = [&]() {
            cublas_gemv(cublasHandle.get(), minfo, salpha, ds_matrix.data(),
                        vinfo, ds_b.data(), sbeta, ds_res.data());
        };
        // ar_type d_error{};
        [[gnu::unused, maybe_unused]] value_type s_error{};
        //[[ gnu::unused, maybe_unused ]] ar_type avv_error{};
        [[gnu::unused, maybe_unused]] value_type avs_error{};
        [[gnu::unused, maybe_unused]] value_type cv_error{};
        [[gnu::unused, maybe_unused]] value_type cs_error{};

        // control_gemv(minfo, v_matrix, vinfo, v_b, v_res_ref);

        d_time = benchmark_function(d_func);
        // d_func();
        // dv_res.get_vector(v_res);
        // d_error = compare(vinfo, v_res_ref, v_res, v_reduce);
        // v_res_ref = v_res;

        s_time = benchmark_function(s_func);
        // ds_res.get_vector(s_res);
        //*
        // std::cout << "single: x_res[0] = " << s_res[0] << '\n';
        // s_error = compare(vinfo, v_res_ref, s_res, v_reduce);
        /*/
        convert_mtx(vinfo, s_res, v_reduce, [](st_type v) { return v.e; });
        s_error = reduce<value_type>(
            vinfo, v_reduce, [](value_type a, value_type b) { return a + b; });
        //*/

        avv_time = benchmark_function(avv_func);
        avs_time = benchmark_function(avs_func);
        // ds_res.get_vector(s_res);
        //*
        // std::cout << "access: x_res[0] = " << s_res[0] << '\n';
        // avs_error = compare(vinfo, v_res_ref, s_res, v_reduce);
        /*/
        convert_mtx(vinfo, s_res, v_reduce, [](st_type v) { return v.e; });
        avs_error = reduce<value_type>(
            vinfo, v_reduce, [](value_type a, value_type b) { return a + b; });
        //*/

        cd_time = benchmark_function(cd_func);
        // dv_res.get_vector(v_res);
        // auto cd_error = compare(vinfo, v_res_ref, v_res, v_reduce);
        // std::cout << cd_error << '\n';
        // std::cout << s_error << ' ' << avs_error << '\t'
        //          << (s_error / avs_error) << '\n';
        cs_time = benchmark_function(cs_func);
        // ds_res.get_vector(s_res);
        //*
        // std::cout << "access: x_res[0] = " << s_res[0] << '\n';
        // cs_error = compare(vinfo, v_res_ref, s_res, v_reduce);

        //*
        std::cout << num_rows << DELIM << d_time << DELIM << s_time << DELIM
                  << avv_time << DELIM << avs_time << DELIM << cd_time << DELIM
                  << cs_time << '\n';
        //*/
        /*
        std::cout << "Comparison:"
                << "\nDouble: " << d_error
                << "\nSingle: " << s_error
                << "\nAcc_vv: " << avv_error
                << "\nAcc_vs: " << avs_error
                << '\n';
        //*/
    }
}
