#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>
#include <functional>
#include <iomanip>
#include <ios>
#include <iostream>
#include <random>
#include <type_traits>
#include <vector>


#include "dot_kernels.cuh"
#include "dot_memory.cuh"
#include "memory.cuh"
#include "utils.cuh"


int main(int argc, char **argv)
{
    using ar_type = double;
    using st_type = float;
    using value_type = ar_type;

    constexpr std::size_t max_size{535 * 1000 * 1000};
    constexpr char DELIM{';'};

    bool detailed_error{false};

    const std::string use_error_string("--error");
    if (argc == 2 && std::string(argv[1]) == use_error_string) {
        detailed_error = true;
    } else if (argc > 1) {
        const std::string binary(argv[0]);
        std::cerr << "Unsupported parameters!\n";
        std::cerr << "Usage: " << binary << " [" << use_error_string << "]\n";
        std::cerr << "With " << use_error_string
                  << ":    compute detailed error of DOTs\n"
                  << "Without parameters: benchmark different DOTs\n";
        return 1;
    }
    std::default_random_engine rengine(42);
    std::uniform_real_distribution<value_type> vector_dist(0.0, 1.0);

    auto ar_data = DotMemory<ar_type>(max_size, vector_dist, rengine);
    auto st_data = DotMemory<st_type>(ar_data);

    auto cublas_handle = cublas_get_handle();
    cublas_set_device_ptr_mode(cublas_handle.get());

    auto my_handle = std::make_unique<myBlasHandle>();

    auto ar_get_result = [&ar_data]() { return ar_data.get_result(); };
    auto st_get_result = [&st_data]() {
        return static_cast<ar_type>(st_data.get_result());
    };

    constexpr std::size_t benchmark_reference{0};
    using benchmark_info_t =
        std::tuple<std::string, std::function<void(matrix_info, matrix_info)>,
                   std::function<value_type()>>;
    std::vector<benchmark_info_t> benchmark_info = {
        benchmark_info_t{"DOT fp64",
                         [&](matrix_info x_info, matrix_info y_info) {
                             dot(my_handle.get(), x_info, ar_data.gpu_x(), y_info,
                                 ar_data.gpu_y(), ar_data.gpu_res());
                         },
                         ar_get_result},
        benchmark_info_t{"DOT fp32",
                         [&](matrix_info x_info, matrix_info y_info) {
                             dot(my_handle.get(), x_info, st_data.gpu_x(), y_info,
                                 st_data.gpu_y(), st_data.gpu_res());
                         },
                         st_get_result},
        benchmark_info_t{"DOT Acc<fp64, fp64>",
                         [&](matrix_info x_info, matrix_info y_info) {
                             acc_dot<double>(
                                 my_handle.get(), x_info, ar_data.gpu_x(), y_info,
                                 ar_data.gpu_y(), ar_data.gpu_res());
                         },
                         ar_get_result},
        benchmark_info_t{"DOT Acc<fp64, fp32>",
                         [&](matrix_info x_info, matrix_info y_info) {
                             acc_dot<double>(
                                 my_handle.get(), x_info, st_data.gpu_x(), y_info,
                                 st_data.gpu_y(), st_data.gpu_res());
                         },
                         st_get_result},
        benchmark_info_t{"DOT Acc<fp32, fp32>",
                         [&](matrix_info x_info, matrix_info y_info) {
                             acc_dot<float>(my_handle.get(), x_info,
                                            st_data.gpu_x(), y_info,
                                            st_data.gpu_y(), st_data.gpu_res());
                         },
                         st_get_result},
        benchmark_info_t{"CUBLAS DOT fp64",
                         [&](matrix_info x_info, matrix_info y_info) {
                             cublas_dot(cublas_handle.get(), x_info,
                                        ar_data.gpu_x(), y_info,
                                        ar_data.gpu_y(), ar_data.gpu_res());
                         },
                         ar_get_result},
        benchmark_info_t{"CUBLAS DOT fp32",
                         [&](matrix_info x_info, matrix_info y_info) {
                             cublas_dot(cublas_handle.get(), x_info,
                                        st_data.gpu_x(), y_info,
                                        st_data.gpu_y(), st_data.gpu_res());
                         },
                         st_get_result}};
    const std::size_t benchmark_num{benchmark_info.size()};

    std::cout << "Distribution vector: [" << vector_dist.a() << ','
              << vector_dist.b() << ")\n";


    std::cout << "Vector Size";
    if (!detailed_error) {
        for (const auto &info : benchmark_info) {
            std::cout << DELIM << std::get<0>(info);
        }
        for (const auto &info : benchmark_info) {
            std::cout << DELIM << "Error " << std::get<0>(info);
        }
    } else {
        for (const auto &info : benchmark_info) {
            std::cout << DELIM << "Error " << std::get<0>(info);
        }
    }
    std::cout << '\n';

    std::cout.precision(16);
    std::cout << std::scientific;

    auto get_error = [](value_type res, value_type ref_res) -> value_type {
        return std::abs(res - ref_res) / std::abs(ref_res);
    };

    // Number of elements of a vector at the start of the benchmark
    constexpr std::size_t start = std::min(max_size, std::size_t{1'000'000});
    // Increase in number of elements between consecutive benchmark runs
    constexpr std::size_t row_incr = 2'000'000;
    // Number of benchmark runs (ignoring randomization)
    constexpr std::size_t steps = (max_size - start) / row_incr;
    // Number of benchmark restarts with a different randomization for vectors
    // Only used for a detailed error run
    constexpr std::size_t randomize_num{10};

    std::vector<std::size_t> benchmark_vec_size((steps + 1));
    std::vector<double> benchmark_time((steps + 1) * benchmark_num);
    // std::vector<value_type> benchmark_error((steps + 1) * benchmark_num);
    // stores the result for all different benchmark runs to compute the error
    const auto actual_randomize_num = detailed_error ? randomize_num : 1;
    std::vector<value_type> raw_result(actual_randomize_num * (steps + 1) *
                                       benchmark_num);
    const auto get_raw_idx = [benchmark_num, actual_randomize_num](
                                 std::size_t rnd, std::size_t step,
                                 std::size_t bi) {
        return step * actual_randomize_num * benchmark_num +
               bi * actual_randomize_num + rnd;
    };

    for (std::size_t randomize = 0;
         (detailed_error && randomize < randomize_num) ||
         (!detailed_error && randomize < 1);
         ++randomize) {
        if (randomize != 0) {
            write_random({{max_size, 1}}, vector_dist, rengine,
                         ar_data.cpu_x_nc());
            write_random({{max_size, 1}}, vector_dist, rengine,
                         ar_data.cpu_y_nc());
            ar_data.copy_cpu_to_gpu();
            st_data.convert_from(ar_data);
        }
        for (std::size_t vec_size = start, i = 0; vec_size <= max_size;
             vec_size += row_incr, ++i) {
            benchmark_vec_size.at(i) = vec_size;
            const matrix_info x_info{{vec_size, 1}};
            const matrix_info y_info{{vec_size, 1}};

            for (std::size_t bi = 0; bi < benchmark_num; ++bi) {
                const std::size_t idx = i * benchmark_num + bi;
                auto curr_lambda = [&]() {
                    std::get<1>(benchmark_info[bi])(x_info, y_info);
                };
                benchmark_time.at(idx) =
                    benchmark_function(curr_lambda, detailed_error);
                raw_result[get_raw_idx(randomize, i, bi)] =
                    std::get<2>(benchmark_info[bi])();
            }
            // const auto result_ref =
            //    raw_result[get_raw_idx(randomize, i, benchmark_reference)];
            // for (std::size_t bi = 0; bi < benchmark_num; ++bi) {
            //    const std::size_t idx = i * benchmark_num + bi;
            //    benchmark_error.at(idx) +=
            //        get_error(raw_result[get_raw_idx(bi)], result_ref);
            //}
        }
    }
    for (std::size_t i = 0; i <= steps; ++i) {
        if (!detailed_error) {
            std::cout << benchmark_vec_size[i];
            for (std::size_t bi = 0; bi < benchmark_num; ++bi) {
                std::cout << DELIM << benchmark_time[i * benchmark_num + bi];
            }
            const auto result_ref =
                raw_result[get_raw_idx(0, i, benchmark_reference)];
            for (std::size_t bi = 0; bi < benchmark_num; ++bi) {
                std::cout << DELIM
                          << get_error(raw_result[i * benchmark_num + bi],
                                       result_ref);
            }
            std::cout << '\n';
        } else {
            std::cout << benchmark_vec_size[i];
            for (std::size_t bi = 0; bi < benchmark_num; ++bi) {
                // sort and compute the median
                std::array<value_type, randomize_num> local_error;
                for (std::size_t rnd = 0; rnd < randomize_num; ++rnd) {
                    const auto result_ref =
                        raw_result[get_raw_idx(rnd, i, benchmark_reference)];
                    local_error[rnd] = get_error(
                        raw_result[get_raw_idx(rnd, i, bi)], result_ref);
                }
                std::sort(local_error.begin(), local_error.end());
                value_type median{};
                if (randomize_num % 2 == 1) {
                    median = local_error[randomize_num / 2];
                } else {
                    const auto begin_middle = randomize_num / 2 - 1;
                    median = (local_error[begin_middle] +
                              local_error[begin_middle + 1]) /
                             2.0;
                }
                std::cout << DELIM << median;
            }
            std::cout << '\n';
        }
    }
    if (!detailed_error) {
        return 0;
    }
    std::cout << "--------------------------------------------------\n";
    std::cout << "Random iter" << DELIM << "Vector Size";
    for (const auto &info : benchmark_info) {
        std::cout << DELIM << "Result" << std::get<0>(info);
    }
    std::cout << '\n';
    for (std::size_t i = 0; i <= steps; ++i) {
        for (std::size_t randomize = 0; randomize < randomize_num;
             ++randomize) {
            std::cout << randomize << DELIM << benchmark_vec_size[i];
            for (std::size_t bi = 0; bi < benchmark_num; ++bi) {
                std::cout << DELIM << raw_result[get_raw_idx(randomize, i, bi)];
            }
            std::cout << '\n';
        }
    }
}
